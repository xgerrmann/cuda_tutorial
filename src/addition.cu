// Tutorial from Nvidia:
// https://www.nvidia.com/docs/IO/116711/sc11-cuda-c-basics.pdf
// Works with CUDA 7.5


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>


__global__ void add(int *a, int *b, int *c){
	*c = *a + *b;
}


int main(void){
	int a, b, c;
	int *d_a, *d_b, *d_c;
	int size = sizeof(int);

	hipMalloc((void**)&d_a, size);
	hipMalloc((void**)&d_b, size);
	hipMalloc((void**)&d_c, size);

	a = 2;
	b = 7;

	hipMemcpy(d_a, &a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, &b, size, hipMemcpyHostToDevice);

	add<<<1,1>>>(d_a, d_b, d_c);

	hipMemcpy(&c, d_c, size, hipMemcpyDeviceToHost);

	hipFree(d_a); hipFree(d_b); hipFree(d_c);

	std::cerr << "2 + 7 = " << c << std::endl;

	return 0;
}
