
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <iostream>
#include <time.h>	// for the seed of the random number generator
__global__ void add(int *a, int *b, int *c){
	c[blockIdx.x] = a[blockIdx.x] + b[blockIdx.x];
}

// CPU function to generate a vector of random integers
// Source:
// https://docs.it4i.cz/anselm-cluster-documentation/software/nvidia-cuda
void random_ints (int *a, int n) {
	srand(time(NULL)); // Set a random seed (unique if execured once per second)
	for (int i = 0; i < n; i++)
	a[i] = rand() % 10000; // random number between 0 and 9999
}

#define N 512

int main(void){
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);

	hipMalloc((void **)&d_a, size);
	hipMalloc((void **)&d_b, size);
	hipMalloc((void **)&d_c, size);

	a = (int *)malloc(size); random_ints(a,N);
	b = (int *)malloc(size); random_ints(b,N);
	c = (int *)malloc(size);

	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

	add<<<N,1>>>(d_a, d_b, d_c);

	hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

	std::cerr << "c[0]: " << c[0] << std::endl;
	std::cerr << "c[1]: " << c[1] << std::endl;

	free(a); free(b); free(c);
	hipFree(d_a); hipFree(d_b); hipFree(d_c);
	return 0;
}
